
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello world from the GPU\n");
}

int main(void)
{
    hello_from_gpu<<<4,4>>>();
    hipDeviceSynchronize();

    return 0;
}